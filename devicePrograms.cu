#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <hip/hip_runtime.h>
#include "gdt/random/random.h"
#include "Interaction.h"

using namespace osc;

namespace osc {
    enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };

    typedef gdt::LCG<16> Random;

    struct PRD {
        Random random;
        vec3f pixelColor;
        vec3f pixelNormal;
        vec3f pixelAlbedo;
    };

    static __forceinline__ __device__
    void *unpackPointer(uint32_t i0, uint32_t i1) {
        const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
        void *ptr = reinterpret_cast<void *>(uptr);
        return ptr;
    }

    static __forceinline__ __device__
    void packPointer(void *ptr, uint32_t &i0, uint32_t &i1) {
        const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
        i0 = uptr >> 32;
        i1 = uptr & 0x00000000ffffffff;
    }

    template<typename T>
    static __forceinline__ __device__ T *getPRD() {
        const uint32_t u0 = optixGetPayload_0();
        const uint32_t u1 = optixGetPayload_1();
        return reinterpret_cast<T *>(unpackPointer(u0, u1));
    }

    extern "C" __constant__ LaunchParams optixLaunchParams;

    extern "C" __global__ void __closesthit__radiance() {
        const TriangleMeshSBTData &sbtData = *(const TriangleMeshSBTData *) optixGetSbtDataPointer();
        Interaction &prd = *(Interaction *) getPRD<Interaction>();
        const int primID = optixGetPrimitiveIndex();
        const vec3i index = sbtData.index[primID];
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;
        const vec3f &A = sbtData.vertex[index.x];
        const vec3f &B = sbtData.vertex[index.y];
        const vec3f &C = sbtData.vertex[index.z];

        vec3f N;
        if (sbtData.normal) {
            N = (1 - u - v) * sbtData.normal[index.x] + u * sbtData.normal[index.y] + v * sbtData.normal[index.z];
            if (N == vec3f(0.f)) {
                N = cross(B - A, C - A);
            }
        } else {
            N = cross(B - A, C - A);
        }

        N = normalize(N);


        const vec3f rayDir = optixGetWorldRayDirection();
        if (dot(N, rayDir) > 0) N = -N;

        prd.position = (1 - u - v) * A + u * B + v * C;
        prd.geoNormal = N;
        vec3f diffuseColor = sbtData.color;

        if (sbtData.hasTexture && sbtData.texcoord) {
            const vec2f tc = (1 - u - v) * sbtData.texcoord[index.x] + u * sbtData.texcoord[index.y] + v * sbtData.
                             texcoord[index.z];
            vec4f fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
            diffuseColor *= (vec3f) fromTexture;
        }

        float cosDN = 0.2f + .8f * fabsf(dot(rayDir, N));
        prd.mat_color = cosDN * diffuseColor;
    }

    extern "C" __global__ void __anyhit__radiance() {
    }

    extern "C" __global__ void __miss__radiance() {
        Interaction &isec = *(Interaction *) getPRD<Interaction>();
        isec.mat_color = vec3f(1.0);
        isec.distance = FLT_MAX;
    }

    extern "C" __global__ void __raygen__renderFrame() {
        const int ix = optixGetLaunchIndex().x;
        const int iy = optixGetLaunchIndex().y;

        const auto &camera = optixLaunchParams.camera;

        const int numPixelSamples = optixLaunchParams.numPixelSamples;

        vec3f pixelColor = 0.f;
        PRD prd;
        prd.random.init(ix + optixLaunchParams.frame.size.x * iy, optixLaunchParams.frame.frameID);

        const vec2f screen(vec2f(ix + 0.5f, iy + 0.5f) / vec2f(optixLaunchParams.frame.size));

        vec3f rayDir = normalize(
            camera.direction + (screen.x - 0.5f) * camera.horizontal + (screen.y - 0.5f) * camera.vertical);


        for (int sampleID = 0; sampleID < numPixelSamples; sampleID++) {
            Ray ray;
            ray.origin = camera.position;
            ray.direction = rayDir;
            vec3f radiance = 0.0f;
            vec3f accum = 1.0f;

            Interaction isect; // 不能放外面
            for (int bounce = 0;; bounce++) {
                if (bounce >= optixLaunchParams.maxBounce) {
                    //radiance = 0;
                    break;
                }
                uint32_t u0, u1;
                packPointer(&isect, u0, u1);
                optixTrace(optixLaunchParams.traversable,
                           ray.origin,
                           ray.direction,
                           0.f, // tmin
                           1e20f, // tmax
                           0.0f, // rayTime
                           OptixVisibilityMask(255),
                           OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
                           SURFACE_RAY_TYPE, // SBT offset
                           RAY_TYPE_COUNT, // SBT stride
                           SURFACE_RAY_TYPE, // missSBTIndex
                           u0, u1);
                if (isect.distance == FLT_MAX) {
                    radiance += vec3f(1.0f) * accum;
                    break;
                }
                radiance += 0;
                accum *= isect.mat_color;
                vec3f wi;
                vec3f rnd;
                rnd.x = prd.random() * 2 - 1;
                rnd.y = prd.random() * 2 - 1;
                rnd.z = prd.random() * 2 - 1;
                wi = normalize(isect.geoNormal + normalize(rnd));
                ray = isect.spawnRay(wi);
                //printf("sample %d: radiance = %.2f %.2f %.2f\n", sampleID, radiance.x, radiance.y, radiance.z);
            }
            pixelColor += radiance;
        }
        vec4f rgba(pixelColor / numPixelSamples, 1.f);
        rgba.x = powf(rgba.x, 1 / 2.2f);
        rgba.y = powf(rgba.y, 1 / 2.2f);
        rgba.z = powf(rgba.z, 1 / 2.2f);
        if (rgba.x > 1)rgba.x = 1.0f;
        if (rgba.y > 1)rgba.y = 1.0f;
        if (rgba.z > 1)rgba.z = 1.0f;
        if (rgba.w > 1)rgba.w = 1.0f;

        // and write to frame buffer ...
        const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;
        if (optixLaunchParams.frame.frameID > 0) {
            rgba += vec4f(optixLaunchParams.frame.colorBuffer[fbIndex]) * float(optixLaunchParams.frame.frameID);
            rgba /= optixLaunchParams.frame.frameID + 1.f;
        }
        optixLaunchParams.frame.colorBuffer[fbIndex] = make_float4(rgba.x, rgba.y, rgba.z, rgba.w);
    }
}
